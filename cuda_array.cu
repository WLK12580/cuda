#include <hip/hip_runtime.h>

#include <stdio.h>

#include <array>
#include <iostream>
__global__ void kernel_func(int *arr13, int *arr11, int *arr12) {  //__global__ 表示核函数在GPU上执行
  int idx = threadIdx.x;  //threadIdx指的是线程的索引
  arr13[idx] = arr11[idx] * arr12[idx];
}
int main() {
  int arr1[3] = {1, 3, 5};
  int arr2[3] = {2, 4, 6};
  int arr3[3];
  int *arr11 = nullptr; //device变量
  int *arr12 = nullptr;
  int *arr13 = nullptr;
  hipMalloc((void **)&arr13, sizeof(int) * 3); //给设备变量分配内存
  hipMalloc((void **)&arr11, sizeof(int) * 3);
  hipMalloc((void **)&arr12, sizeof(int) * 3);
  hipMemcpy(arr11, arr1, sizeof(int) * 3, hipMemcpyHostToDevice); //从主机复制数据到GPU
  hipMemcpy(arr12, arr2, sizeof(int) * 3, hipMemcpyHostToDevice);
  hipMemcpy(arr13, arr3, sizeof(int) * 3, hipMemcpyHostToDevice);
  kernel_func<<<1, 3>>>(arr13, arr11, arr12); //分配一个块三个线程，同一个块中的线程是同步的

  hipError_t  cudaStatus = hipMemcpy(arr3, arr13, 3 * sizeof(int), hipMemcpyDeviceToHost); //将计算结果返回到主机
  if(cudaStatus != hipSuccess){
    std::cout<<"error\n";
    return 0;
  }
  for (int i = 0; i < 3; i++) {
    printf("arr13=%d\n", arr3[i]);
  }
  hipFree(arr1);
  hipFree(arr2);
  hipFree(arr3);
  hipDeviceReset();
  return 0;
}
