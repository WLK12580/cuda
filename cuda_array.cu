#include<hip/hip_runtime.h>

#include<iostream>
#include<array>
#include<stdio.h>
__global__ void kernel_func(int *arr3,const int *arr1,const int *arr2){
    printf(" hello GPU\n");
    const unsigned int thread_idx=threadIdx.x;
    arr3[thread_idx]=arr1[thread_idx]*arr2[thread_idx];
    printf("arr1=%d",arr1[thread_idx]);
}
int main(){
    int *arr1[101];
    int *arr2[101];
    int *arr3[101];
    int *arr12[101];
    int *arr11[101];
    int *arr13[101];
    std::cout<<"test01"<<std::endl;
    for(int i=0;i<101;i++){
        std::cout<<"i="<<i<<std::endl;
    }
    std::cout<<"test02"<<std::endl;
    hipMalloc((void**)&arr3,sizeof(int)*101);
    hipMalloc((void**)&arr1,sizeof(int)*101);
    hipMalloc((void**)&arr2,sizeof(int)*101);
    
    hipMemcpy(arr1,arr11,101*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(arr2,arr12,101*sizeof(int),hipMemcpyHostToDevice);
    kernel_func<<<1,100>>>(*arr3,*arr1,*arr2);
    hipMemcpy(arr13,arr3,101*sizeof(int),hipMemcpyDeviceToHost);

    hipFree(arr1);
    hipFree(arr2);
    hipFree(arr3);
    hipDeviceReset();
    for(int i=0;i<sizeof(arr13)/sizeof(int);i++){
        std::cout<<" arr["<<i<<"]="<<arr13[i];
    }
    std::cout<<std::endl;

    return 0;
}

