#include <hip/hip_runtime.h>

#include<stdio.h>
__global__ void helloGPU(){
    printf(" hello GPU\n");
}
int main(){
    // cudaError_t cudastatus;
    printf("hello cpu\n");
    helloGPU<<<1,10>>>();
    hipDeviceReset();
    return 0;
}